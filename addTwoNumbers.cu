// Kylo Ren c2022
#include<iostream>
#include<hip/hip_runtime.h>

using namespace std;

__global__ void addInteger(int* a, int* b){
    a[0] += b[0];
}

int main(){
    int a=2, b=3;
    int *d_a, *d_b;

    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    addInteger<<< 1, 1 >>>(d_a, d_b);

    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

    cout<<"Sum of "<<a<<" and "<<b<<" is : "<< a<<endl;

    return 0;
}

